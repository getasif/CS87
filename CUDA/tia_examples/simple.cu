#include "hip/hip_runtime.h"
// a simple example of how to use myopengllib library to 
// animate a cuda computation
// Most of the openGL-Cuda interoperability stuff is hidden
// in the GPUDisplayData class.  However, you need to think
// a little bit about writing an event driven program since
// you register animation and clean-up events with this libary
// and then run your animation.
//
// This example application doesn't do anything recognizable,
// but shows how to use the GPUDisplayData library and how to
// write animate and clean_up functions to pass to the  
// GPUDisplayData.AnimateComputation  method.
//
// (newhall, 2011)

#include "myopengllib.h"


#define DIM 300

static void animate_simple(uchar4 *disp, void *mycudadata);
static void clean_up(void);
__global__ void int_to_color( uchar4 *optr, const int *my_cuda_data );
__global__ void  simplekernel(int *data, int* b); 


// just use a global to avoid huge stack space usage
// of a local (mallocing up heap space would be fine too)
static int matrix[DIM][DIM];

// if your program needs more GPU data, use a struct
// with fields for each value needed.
typedef struct my_cuda_data {
  int *dev_grid;
  int *bogus;
  int start;
  float end;

} my_cuda_data;

// single var holds all program data.  This will be passed
// to the GPUDisplayData constructor 
my_cuda_data simple_prog_data;

int main(int argc, char *argv[])  {


  // The call to the constructor has to come before any calls to 
  // hipMalloc or other Cuda routines
  // This is part of the reason why we are passing the address of 
  // a struct with fields which are ptrs to hipMalloc'ed space
  // The other reason is that adding a level of interection 
  // is the answer to every problem.
  GPUDisplayData my_display(DIM, DIM, &simple_prog_data, "Simple openGL-Cuda");

  // initialize application data 
  for(int i =0; i < DIM; i++) {
    for(int j =0; j < DIM; j++) {
      matrix[i][j] = j;
    }
  }

  // allocate memory space for our application data on the GPU
  HANDLE_ERROR(hipMalloc((void**)&simple_prog_data.dev_grid, 
        sizeof(int)*DIM*DIM), "malloc dev_grid") ;

  // copy the initial data to the GPU
  HANDLE_ERROR (hipMemcpy(simple_prog_data.dev_grid, matrix, 
        sizeof(int)*DIM*DIM, hipMemcpyHostToDevice), "copy dev_grid to GPU") ;

  HANDLE_ERROR(hipMalloc((void**)&simple_prog_data.bogus, 
        sizeof(int)*DIM*DIM), "malloc dev_grid") ;


  // register a clean-up function on exit that will call hipFree 
  // on any hipMalloc'ed space
  my_display.RegisterExitFunction(clean_up); 

  // have the library run our Cuda animation
  my_display.AnimateComputation(animate_simple);

  return 0;
}

// cleanup function passed to AnimateComputatin method.
// it is called when the program exits and should clean up
// all hipMalloc'ed state.
// Your clean-up function's prototype must match this, which is 
// why simple_prog_data needs to be a global
static void clean_up(void) {
  hipFree(simple_prog_data.dev_grid);
  hipFree(simple_prog_data.bogus);
}

// amimate function passed to AnimateComputation: 
// this function will be called by openGL's dislplay function.
// It can contain code that runs on the CPU and also calls to
// to CUDA kernel code to do a computation and to change the
// display the results using openGL...you need to change the
// display color values based on the application values
// 
// devPtr: is pointer into openGL buffer of rgba values (but
//         the field names are x,y,z,w
// my_data: is pointer to our cuda data that we passed into the 
//          constructor
// 
// your animate function prototype must match this one:
static void animate_simple(uchar4 *devPtr, void *my_data) {

  my_cuda_data *simple_data = (my_cuda_data *)my_data;
  dim3 blocks(DIM, DIM); 

  // comment out the for loop to do a display update every 
  // execution of simplekernel
  //for(int i=0; i < 90; i++) 
    simplekernel<<<blocks,1>>>( simple_data->dev_grid, simple_data->bogus); 

  int_to_color<<<blocks,1>>>(devPtr, simple_data->dev_grid); 
}

// a kernel to set the color the opengGL display object based 
// on the cuda data value
//  
//  optr: is an array of openGL RGB pixels, each is a 
//        4-tuple (x:red, y:green, z:blue, w:opacity) 
//  my_cuda_data: is cuda 2D array of ints
__global__ void int_to_color( uchar4 *optr, const int *my_cuda_data ) {

    // get this thread's block position to map into
    // location in opt and my_cuda_data
    // the row and col values depend on how you parallelize the
    // kernel (<<<blocks, threads>>>).  This is how to do it for
    // a DIMxDIM grid of blocks, each block with a single thread
    // If each block has more than one thread, or blocks of threads
    // then you need to do something more complicated to get a 
    // thread's mapping to a row and col value
    int row = blockIdx.x;  
    int col = blockIdx.y;
    int offset = col + row * gridDim.x;

    if(col < DIM && row < DIM) {
      // change this pixel's color value based on some strange
      // functions of the my_cuda_data value
      optr[offset].x = (my_cuda_data[offset]+10)%255;  // R value
      optr[offset].y = (my_cuda_data[offset]+100)%255; // G value
      optr[offset].z = (my_cuda_data[offset]+200)%255; // B value
      optr[offset].w = 255;  // just set this to 255 always
    }
}

// a simple cuda kernel: cyclicly increases a points value by 10
//  data: a "2D" array of int values
__global__ void  simplekernel(int *data, int* b) {

    int row = blockIdx.x;
    int col = blockIdx.y;
    int offset = col + row * gridDim.y;

    if(col < DIM && row < DIM) {
      data[offset] = (data[offset]+10)%1000;
    }
    b[offset] = 1;




}


