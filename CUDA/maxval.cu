#include "hip/hip_runtime.h"
#include "book.h"
#include <climits>
#include <cstdlib>
#include <iostream>
#include "timer.h"

using namespace std;

#define imin(a,b) (a<b?a:b)

const int N = 16*1024*1024;

/* return a random floating point value in the range min to max */
float rand_in_range(float min, float max){
  return (max-min)*(1.0*(rand()%INT_MAX))/INT_MAX+min;
}

/* N is a global constant */
float max_cpu(float *a, int size){
  float mval = a[0];
  for(int i=1; i<size; i++){
    if (a[i] > mval){
      mval = a[i];
    }
  }
  return mval;
}

__global__ void max_gpu_single(float* dev_a, int size, float* dev_result ){
  float lmax=dev_a[0];
	for(int i=0; i<size; i++){
		if(dev_a[i] > lmax){
			lmax = dev_a[i];
		}
	}
  dev_result[0]=lmax;
}

int block_test(int numBlocks, float* dev_a, int size, float* dev_result){
	printf("Block kernel with %d blocks and 1 thread per block\n", numBlocks);
	//max_gpu_block<<<numBlocks,1>>>(dev_a,size,dev_result);
  return numBlocks;
}

int thread_test(int numThreads, float* dev_a, int size, float* dev_result){
	printf("Thread kernel with 1 block and %d threads per block\n", numThreads);
	//max_gpu_thread<<<1,numThreads>>>(dev_a,size,dev_result);
  return numThreads;
}

int combo_test(int numBlocks, int numThreads,
		float* dev_a, int size, float* dev_result){
	printf("Combo kernel with %d blocks and %d threads per block\n", 
			numBlocks, numThreads);
	//max_gpu_combined<<<numBlocks,numThreads>>>(dev_a,size,dev_result);
  return numBlocks;
}

void time_kernel(GPUTimer& gtime, float* dev_a, int size, float* dev_result,
		float* partial_result, int testID){

    int partial_size;
		gtime.start();

    // call the kernel
		switch (testID){
			case 1:
        partial_size=1;
        max_gpu_single<<<1,1>>>(dev_a,size,dev_result);
				break;
		  default:
				break;
		}

		gtime.stop();
		gtime.print();

    // copy the array 'result' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( partial_result, dev_result,
                              partial_size*sizeof(float),
                              hipMemcpyDeviceToHost ) );

    CPUTimer t;
    t.start();
    
    // finish up on the CPU side
    float ans = partial_result[0];
		for (int i=0; i<partial_size; i++) {
			if(partial_result[i] > ans){
				ans = partial_result[i];
			}
		}

    cout << "Max (by GPU) " << ans << endl;
    t.stop();
    printf("Time to run CPU-GPU finishing: %7.2f ms\n", 1000*t.elapsed());
		printf("\n");

}

int main( void ) {
    float   *a, *partial_result;
    float   *dev_a, *dev_result;
   
    GPUTimer gtime;

    //MAX size of partial results buffer
    int partial_size = 2048;

    // allocate memory on the cpu side
    a = (float*)malloc( N*sizeof(float) );
    partial_result = (float*)malloc( partial_size*sizeof(float) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a,
                              N*sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_result,
                              partial_size*sizeof(float) ) );

    // fill in the host memory with data
    for (int i=0; i<N; i++) {
        a[i] = rand_in_range(0.0f,50000.0f);
        //cout << i << " " << a[i] << endl;
    }

    //initialize partial results to be all 0
    for(int i=0; i<partial_size; i++){
      partial_result[i]=0.0f;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N*sizeof(float),
                              hipMemcpyHostToDevice ) );

    CPUTimer t;
    t.start();
    cout << "Max (by CPU) " << max_cpu(a, N) << endl;
    t.stop();
    printf("Time to run on CPU: %7.2f ms\n\n", 1000*t.elapsed());

		for( int t=1; t<2; t++){
		  time_kernel(gtime, dev_a, N, dev_result, partial_result, t);
    }

		// free memory on the gpu side
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_result ) );

    // free memory on the cpu side
    free( a ); a=NULL;
    // free memory on the cpu side
    free( partial_result ); partial_result=NULL;
}
